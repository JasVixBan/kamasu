#include "hip/hip_runtime.h"
#include "kernel_util.hpp"

__global__ void
kamasu_linspace_knl(float* data,
		    float value,
		    unsigned linear_size,
		    const int stride)
{
  if (INDEX >= linear_size)
    return;

  unsigned actual_index =  INDEX*stride;

  data[actual_index] = value;
}

void 
kamasu_assign(float* data, 
	      float value,
	      std::size_t linear_size,
	      const int stride)
{
  bd_t bd = gridsize(linear_size);

  kamasu_linspace_knl<<<bd.first, bd.second>>>
    (data,
     value,
     linear_size,
     stride);
}

