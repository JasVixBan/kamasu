#include "hip/hip_runtime.h"
#include "kernel_util.hpp"

namespace resophonic
{
  namespace kamasu
  {
    __global__ void
    assign_knl(float* data,
	       float value,
	       unsigned linear_size,
	       const int stride)
    {
      if (INDEX >= linear_size)
	return;
      unsigned actual_index =  INDEX*stride;

      data[actual_index] = value;
    }

    void 
    assign(float* data, 
	   float value,
	   std::size_t linear_size,
	   const int stride)
    {
      float* thingy = 0;
      *thingy = 1.0f;

      bd_t bd = gridsize(linear_size);

      assign_knl<<<bd.first, bd.second>>>(data,
					  value,
					  linear_size,
					  stride);
    }

  }
}
