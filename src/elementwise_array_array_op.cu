#include "hip/hip_runtime.h"

#include "elementwise_array_scalar_op.hpp"
#include "kernel_util.hpp"

#include <proto_tags_fwd.hpp>
#include <resophonic/kamasu/tag.hpp>
#include <stdexcept>

namespace resophonic {
  namespace kamasu {

    template <typename T>
    __device__ void 
    op_impl(T* l, T* r, ::boost::proto::tag::plus)
    {
      *l += *r;
    }

    template <typename T>
    __device__ void 
    op_impl(T* l, T* r, ::boost::proto::tag::minus)
    {
      (*l) -= *r;
    }

    template <typename T>
    __device__ void 
    op_impl(T* l, T* r, ::boost::proto::tag::multiplies)
    {
      (*l) *= *r;
    }

    template <typename T>
    __device__ void 
    op_impl(T* l, T* r, ::boost::proto::tag::divides)
    {
      (*l) /= *r;
    }

    template <typename T, int N, typename Tag>
    __global__ void 
    eaa_knl(std::size_t linear_size,
	    T* data_l,
	    T* data_r,
	    argpack<std::size_t, N> factors_l,
	    argpack<std::size_t, N> factors_r,
	    argpack<int, N> strides_l,
	    argpack<int, N> strides_r)
    {
      if (INDEX >= linear_size)
	return;

      unsigned lhs_off = actual_index<N>(factors_l, strides_l);
      unsigned rhs_off = actual_index<N>(factors_r, strides_r);

      op_impl(data_l + lhs_off, data_r + rhs_off, Tag()); 
    }

    template <typename T, int N, typename Tag>
    void 
    elementwise_array_array_op(std::size_t linear_size,
			       T* data_l,
			       T* data_r,
			       const std::size_t* factors_l,
			       const std::size_t* factors_r,
			       const int* strides_l,
			       const int* strides_r,
			       hipStream_t stream)
    {
      bd_t bd = gridsize(linear_size);
      
      argpack<std::size_t, N> factors_l_(factors_l);
      argpack<int, N> strides_l_(strides_l);
      
      argpack<std::size_t, N> factors_r_(factors_r);
      argpack<int, N> strides_r_(strides_r);
      
      eaa_knl<T, N, Tag><<<bd.first, bd.second, 0, stream>>>(linear_size, 
							     data_l, data_r, 
							     factors_l_, factors_r_,
							     strides_l_, strides_r_);
    }

    namespace inst 
    {
      template <typename T, typename Tag>
      struct iaao
      {
	iaao()
	{
	  hipStream_t s;
	  elementwise_array_array_op<T, 1, Tag>(0U, 0, 0, 0, 0, 0, 0, s);
	  elementwise_array_array_op<T, 2, Tag>(0, 0, 0, 0, 0, 0, 0, s);
	  elementwise_array_array_op<T, 3, Tag>(0, 0, 0, 0, 0, 0, 0, s);
	  elementwise_array_array_op<T, 4, Tag>(0, 0, 0, 0, 0, 0, 0, s);
	  elementwise_array_array_op<T, 5, Tag>(0, 0, 0, 0, 0, 0, 0, s);
	}
      };

      template struct iaao<float, boost::proto::tag::plus>;
      template struct iaao<float, boost::proto::tag::minus>;
      template struct iaao<float, boost::proto::tag::divides>;

    }
    template void elementwise_array_array_op<float, 5, boost::proto::tag::minus>(unsigned long, float*, float*, unsigned long const*, unsigned long const*, int const*, int const*, int);
  }
}

