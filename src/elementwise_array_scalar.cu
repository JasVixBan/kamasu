#include "hip/hip_runtime.h"

#include "elementwise_array_scalar.hpp"
#include "kernel_util.hpp"

#include <proto_tags_fwd.hpp>
#include <resophonic/kamasu/tag.hpp>

namespace resophonic {
  namespace kamasu {

    template <typename T>
    __device__ void 
    op_impl(T* t, const T& scalar, ::boost::proto::tag::plus)
    {
      (*t) += scalar;
    }

    template <typename T>
    __device__ void 
    op_impl(T* t, const T& scalar, ::boost::proto::tag::multiplies)
    {
      (*t) *= scalar;
    }

    template <typename T>
    __device__ void 
    op_impl(T* t, const T& scalar, ::boost::proto::tag::divides)
    {
      (*t) /= scalar;
    }

    template <typename T>
    __device__ void 
    op_impl(T* t, const T& scalar, ::boost::proto::tag::minus)
    {
      (*t) -= scalar;
    }

    template <typename T>
    __device__ void 
    op_impl(T* t, const T& scalar, resophonic::kamasu::tag::pow)
    {
      (*t) = pow(*t, scalar);
    }

    template <typename T, int Length>
    struct argpack
    {
      T data[Length];
      argpack(const T* d)
      {
	memcpy(data, d, Length * sizeof(T));
      }
      T& operator[](unsigned index)
      {
	return data[index];
      }
      const T& operator[](unsigned index) const
      {
	return data[index];
      }
      void touchme() { }

    };

    template <int N>
    __device__ unsigned actual_index(const argpack<std::size_t, N> &factors,
				     const argpack<int, N> &strides)
    {
      unsigned x = INDEX/factors[N-1] * strides[N-1];
							//      for (unsigned i=I-1; i<=0; i++)
#pragma unroll 10
      for (int i = N-1; i>0; i--)
	x += unsigned(INDEX % factors[i]) / factors[i-1] * strides[i-1];

      return x;

    }

    template <typename T, int N, typename Tag>
    __global__ void 
    doit(T* data, 
	 std::size_t linear_size, 
	 argpack<std::size_t, N> factors,
	 argpack<int, N> strides,
	 T scalar)
    {
      if (INDEX >= linear_size)
	return;

      unsigned thisthread_offset = actual_index<N>(factors, strides);

      op_impl(data + thisthread_offset, scalar, Tag()); 
    }

    template <typename T, int N, typename Tag>
    void 
    elementwise_array_scalar_op(T* data, 
				std::size_t linear_size, 
				const std::size_t* factors,
				const int* strides,
				T scalar)
    {
      bd_t bd = gridsize(linear_size);
      
      argpack<std::size_t, N> factors_(factors);
      argpack<int, N> strides_(strides);
      
      doit<T, N, Tag><<<bd.first, bd.second>>>(data, linear_size, factors_, strides_, scalar);
    }

    template <typename T, typename Tag>
    struct instantiate {
      instantiate()
      {
	elementwise_array_scalar_op<T, 1, Tag>(0, 0, 0, 0, 0.0f);
	elementwise_array_scalar_op<T, 2, Tag>(0, 0, 0, 0, 0.0f);
	elementwise_array_scalar_op<T, 3, Tag>(0, 0, 0, 0, 0.0f);
	elementwise_array_scalar_op<T, 4, Tag>(0, 0, 0, 0, 0.0f);
	elementwise_array_scalar_op<T, 5, Tag>(0, 0, 0, 0, 0.0f);
      }
    };

    template struct instantiate<float, boost::proto::tag::plus>;
    template struct instantiate<float, boost::proto::tag::minus>;
    template struct instantiate<float, boost::proto::tag::multiplies>;
    template struct instantiate<float, boost::proto::tag::divides>;
    template struct instantiate<float, resophonic::kamasu::tag::pow>;

  }
}

