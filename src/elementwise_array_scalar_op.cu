#include "hip/hip_runtime.h"

#include "elementwise_array_scalar_op.hpp"
#include "kernel_util.hpp"

#include <proto_tags_fwd.hpp>
#include <resophonic/kamasu/tag.hpp>

namespace resophonic {
  namespace kamasu {

    template <typename T>
    __device__ void 
    op_impl(T* t, const T& scalar, ::boost::proto::tag::plus)
    {
      (*t) += scalar;
    }

    template <typename T>
    __device__ void 
    op_impl(T* t, const T& scalar, ::boost::proto::tag::multiplies)
    {
      (*t) *= scalar;
    }

    template <typename T>
    __device__ void 
    op_impl(T* t, const T& scalar, ::boost::proto::tag::divides)
    {
      (*t) /= scalar;
    }

    template <typename T>
    __device__ void 
    op_impl(T* t, const T& scalar, ::boost::proto::tag::minus)
    {
      (*t) -= scalar;
    }

    template <typename T>
    __device__ void 
    op_impl(T* t, const T& scalar, resophonic::kamasu::tag::pow)
    {
      (*t) = pow(*t, scalar);
    }

    template <typename T, int N, typename Tag>
    __global__ void 
    eas_knl(T* data, 
	 std::size_t linear_size, 
	 argpack<std::size_t, N> factors,
	 argpack<int, N> strides,
	 T scalar)
    {
      if (INDEX >= linear_size)
	return;

      unsigned thisthread_offset = actual_index<N>(factors, strides);

      op_impl(data + thisthread_offset, scalar, Tag()); 
    }

    template <typename T, int N, typename Tag>
    void 
    elementwise_array_scalar_op(T* data, 
				std::size_t linear_size, 
				const std::size_t* factors,
				const int* strides,
				T scalar)
    {
      bd_t bd = gridsize(linear_size);
      
      argpack<std::size_t, N> factors_(factors);
      argpack<int, N> strides_(strides);
      
      eas_knl<T, N, Tag><<<bd.first, bd.second>>>(data, linear_size, factors_, strides_, scalar);
    }

    template <typename T, typename Tag>
    struct instantiate 
    {
      instantiate()
      {
	elementwise_array_scalar_op<T, 1, Tag>(0, 0, 0, 0, 0.0f);
	elementwise_array_scalar_op<T, 2, Tag>(0, 0, 0, 0, 0.0f);
	elementwise_array_scalar_op<T, 3, Tag>(0, 0, 0, 0, 0.0f);
	elementwise_array_scalar_op<T, 4, Tag>(0, 0, 0, 0, 0.0f);
	elementwise_array_scalar_op<T, 5, Tag>(0, 0, 0, 0, 0.0f);
      }
    };

    template struct instantiate<float, boost::proto::tag::plus>;
    template struct instantiate<float, boost::proto::tag::minus>;
    template struct instantiate<float, boost::proto::tag::multiplies>;
    template struct instantiate<float, boost::proto::tag::divides>;
    template struct instantiate<float, resophonic::kamasu::tag::pow>;

  }
}


