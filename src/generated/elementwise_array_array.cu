#include "hip/hip_runtime.h"
//
//
// this is a generated file
//
//

#include <resophonic/kamasu/config.hpp>
#include "kernel.h"

__global__ void
kamasu_elementwise_array_array_knl_1(Op op,
					 std::size_t linear_size,
					 float* data_l,
					 float* data_r,
					 const std::size_t factor_l0,
					 const std::size_t factor_r0,
					 const int stride_l0,
					 const int stride_r0)
{
  if (INDEX >= linear_size)
    return;

  unsigned actual_index_l = INDEX/factor_l0*stride_l0;
  unsigned actual_index_r = INDEX/factor_r0*stride_r0;

  if (op == MULTIPLIES)
    data_l[actual_index_l] *= data_r[actual_index_r];
  else if (op == DIVIDES)
    data_l[actual_index_l] /= data_r[actual_index_r];
  else if (op == PLUS)
    data_l[actual_index_l] += data_r[actual_index_r];
  else if (op == MINUS)
    data_l[actual_index_l] -= data_r[actual_index_r];
}

void 
kamasu_elementwise_array_array_1(Op op,
				     std::size_t linear_size,
				     float* data_l,
				     float* data_r,
				     const std::size_t* factors_l,
				     const std::size_t* factors_r,
				     const int* strides_l,
				     const int* strides_r)
{
  bd_t bd = gridsize(linear_size);
  
  kamasu_elementwise_array_array_knl_1<<<bd.first, bd.second>>>
    (op, 
     linear_size,
     data_l,
     data_r,
     factors_l[0],
     factors_r[0],
     strides_l[0],
     strides_r[0]);
}

#include <resophonic/kamasu/config.hpp>
#include "kernel.h"

__global__ void
kamasu_elementwise_array_array_knl_2(Op op,
					 std::size_t linear_size,
					 float* data_l,
					 float* data_r,
					 const std::size_t factor_l0,const std::size_t factor_l1,
					 const std::size_t factor_r0,const std::size_t factor_r1,
					 const int stride_l0,const int stride_l1,
					 const int stride_r0,const int stride_r1)
{
  if (INDEX >= linear_size)
    return;

  unsigned actual_index_l = INDEX/factor_l1*stride_l1 +  unsigned(INDEX % factor_l1)/factor_l0*stride_l0;
  unsigned actual_index_r = INDEX/factor_r1*stride_r1 +  unsigned(INDEX % factor_r1)/factor_r0*stride_r0;

  if (op == MULTIPLIES)
    data_l[actual_index_l] *= data_r[actual_index_r];
  else if (op == DIVIDES)
    data_l[actual_index_l] /= data_r[actual_index_r];
  else if (op == PLUS)
    data_l[actual_index_l] += data_r[actual_index_r];
  else if (op == MINUS)
    data_l[actual_index_l] -= data_r[actual_index_r];
}

void 
kamasu_elementwise_array_array_2(Op op,
				     std::size_t linear_size,
				     float* data_l,
				     float* data_r,
				     const std::size_t* factors_l,
				     const std::size_t* factors_r,
				     const int* strides_l,
				     const int* strides_r)
{
  bd_t bd = gridsize(linear_size);
  
  kamasu_elementwise_array_array_knl_2<<<bd.first, bd.second>>>
    (op, 
     linear_size,
     data_l,
     data_r,
     factors_l[0],factors_l[1],
     factors_r[0],factors_r[1],
     strides_l[0],strides_l[1],
     strides_r[0],strides_r[1]);
}

#include <resophonic/kamasu/config.hpp>
#include "kernel.h"

__global__ void
kamasu_elementwise_array_array_knl_3(Op op,
					 std::size_t linear_size,
					 float* data_l,
					 float* data_r,
					 const std::size_t factor_l0,const std::size_t factor_l1,const std::size_t factor_l2,
					 const std::size_t factor_r0,const std::size_t factor_r1,const std::size_t factor_r2,
					 const int stride_l0,const int stride_l1,const int stride_l2,
					 const int stride_r0,const int stride_r1,const int stride_r2)
{
  if (INDEX >= linear_size)
    return;

  unsigned actual_index_l = INDEX/factor_l2*stride_l2 +  unsigned(INDEX % factor_l1)/factor_l0*stride_l0 +  unsigned(INDEX % factor_l2)/factor_l1*stride_l1;
  unsigned actual_index_r = INDEX/factor_r2*stride_r2 +  unsigned(INDEX % factor_r1)/factor_r0*stride_r0 +  unsigned(INDEX % factor_r2)/factor_r1*stride_r1;

  if (op == MULTIPLIES)
    data_l[actual_index_l] *= data_r[actual_index_r];
  else if (op == DIVIDES)
    data_l[actual_index_l] /= data_r[actual_index_r];
  else if (op == PLUS)
    data_l[actual_index_l] += data_r[actual_index_r];
  else if (op == MINUS)
    data_l[actual_index_l] -= data_r[actual_index_r];
}

void 
kamasu_elementwise_array_array_3(Op op,
				     std::size_t linear_size,
				     float* data_l,
				     float* data_r,
				     const std::size_t* factors_l,
				     const std::size_t* factors_r,
				     const int* strides_l,
				     const int* strides_r)
{
  bd_t bd = gridsize(linear_size);
  
  kamasu_elementwise_array_array_knl_3<<<bd.first, bd.second>>>
    (op, 
     linear_size,
     data_l,
     data_r,
     factors_l[0],factors_l[1],factors_l[2],
     factors_r[0],factors_r[1],factors_r[2],
     strides_l[0],strides_l[1],strides_l[2],
     strides_r[0],strides_r[1],strides_r[2]);
}

#include <resophonic/kamasu/config.hpp>
#include "kernel.h"

__global__ void
kamasu_elementwise_array_array_knl_4(Op op,
					 std::size_t linear_size,
					 float* data_l,
					 float* data_r,
					 const std::size_t factor_l0,const std::size_t factor_l1,const std::size_t factor_l2,const std::size_t factor_l3,
					 const std::size_t factor_r0,const std::size_t factor_r1,const std::size_t factor_r2,const std::size_t factor_r3,
					 const int stride_l0,const int stride_l1,const int stride_l2,const int stride_l3,
					 const int stride_r0,const int stride_r1,const int stride_r2,const int stride_r3)
{
  if (INDEX >= linear_size)
    return;

  unsigned actual_index_l = INDEX/factor_l3*stride_l3 +  unsigned(INDEX % factor_l1)/factor_l0*stride_l0 +  unsigned(INDEX % factor_l2)/factor_l1*stride_l1 +  unsigned(INDEX % factor_l3)/factor_l2*stride_l2;
  unsigned actual_index_r = INDEX/factor_r3*stride_r3 +  unsigned(INDEX % factor_r1)/factor_r0*stride_r0 +  unsigned(INDEX % factor_r2)/factor_r1*stride_r1 +  unsigned(INDEX % factor_r3)/factor_r2*stride_r2;

  if (op == MULTIPLIES)
    data_l[actual_index_l] *= data_r[actual_index_r];
  else if (op == DIVIDES)
    data_l[actual_index_l] /= data_r[actual_index_r];
  else if (op == PLUS)
    data_l[actual_index_l] += data_r[actual_index_r];
  else if (op == MINUS)
    data_l[actual_index_l] -= data_r[actual_index_r];
}

void 
kamasu_elementwise_array_array_4(Op op,
				     std::size_t linear_size,
				     float* data_l,
				     float* data_r,
				     const std::size_t* factors_l,
				     const std::size_t* factors_r,
				     const int* strides_l,
				     const int* strides_r)
{
  bd_t bd = gridsize(linear_size);
  
  kamasu_elementwise_array_array_knl_4<<<bd.first, bd.second>>>
    (op, 
     linear_size,
     data_l,
     data_r,
     factors_l[0],factors_l[1],factors_l[2],factors_l[3],
     factors_r[0],factors_r[1],factors_r[2],factors_r[3],
     strides_l[0],strides_l[1],strides_l[2],strides_l[3],
     strides_r[0],strides_r[1],strides_r[2],strides_r[3]);
}

#include <resophonic/kamasu/config.hpp>
#include "kernel.h"

__global__ void
kamasu_elementwise_array_array_knl_5(Op op,
					 std::size_t linear_size,
					 float* data_l,
					 float* data_r,
					 const std::size_t factor_l0,const std::size_t factor_l1,const std::size_t factor_l2,const std::size_t factor_l3,const std::size_t factor_l4,
					 const std::size_t factor_r0,const std::size_t factor_r1,const std::size_t factor_r2,const std::size_t factor_r3,const std::size_t factor_r4,
					 const int stride_l0,const int stride_l1,const int stride_l2,const int stride_l3,const int stride_l4,
					 const int stride_r0,const int stride_r1,const int stride_r2,const int stride_r3,const int stride_r4)
{
  if (INDEX >= linear_size)
    return;

  unsigned actual_index_l = INDEX/factor_l4*stride_l4 +  unsigned(INDEX % factor_l1)/factor_l0*stride_l0 +  unsigned(INDEX % factor_l2)/factor_l1*stride_l1 +  unsigned(INDEX % factor_l3)/factor_l2*stride_l2 +  unsigned(INDEX % factor_l4)/factor_l3*stride_l3;
  unsigned actual_index_r = INDEX/factor_r4*stride_r4 +  unsigned(INDEX % factor_r1)/factor_r0*stride_r0 +  unsigned(INDEX % factor_r2)/factor_r1*stride_r1 +  unsigned(INDEX % factor_r3)/factor_r2*stride_r2 +  unsigned(INDEX % factor_r4)/factor_r3*stride_r3;

  if (op == MULTIPLIES)
    data_l[actual_index_l] *= data_r[actual_index_r];
  else if (op == DIVIDES)
    data_l[actual_index_l] /= data_r[actual_index_r];
  else if (op == PLUS)
    data_l[actual_index_l] += data_r[actual_index_r];
  else if (op == MINUS)
    data_l[actual_index_l] -= data_r[actual_index_r];
}

void 
kamasu_elementwise_array_array_5(Op op,
				     std::size_t linear_size,
				     float* data_l,
				     float* data_r,
				     const std::size_t* factors_l,
				     const std::size_t* factors_r,
				     const int* strides_l,
				     const int* strides_r)
{
  bd_t bd = gridsize(linear_size);
  
  kamasu_elementwise_array_array_knl_5<<<bd.first, bd.second>>>
    (op, 
     linear_size,
     data_l,
     data_r,
     factors_l[0],factors_l[1],factors_l[2],factors_l[3],factors_l[4],
     factors_r[0],factors_r[1],factors_r[2],factors_r[3],factors_r[4],
     strides_l[0],strides_l[1],strides_l[2],strides_l[3],strides_l[4],
     strides_r[0],strides_r[1],strides_r[2],strides_r[3],strides_r[4]);
}

