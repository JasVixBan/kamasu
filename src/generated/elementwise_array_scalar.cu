#include "hip/hip_runtime.h"
//
//
// this is a generated file
//
//

#include <resophonic/kamasu/config.hpp>
#include <kernel.h>

// bah bah bah
__global__ void
kamasu_elementwise_array_scalar_1_knl
(Op op,
 float* data,
 unsigned linear_size,
 const std::size_t factor0,
 const int stride0,
 float scalar)
{
  if (INDEX >= linear_size)
    return;

  unsigned actual_index = 
    INDEX/factor0*stride0;

  if (op == MULTIPLIES)
    data[actual_index] *= scalar;
  else if (op == DIVIDES)
    data[actual_index] /= scalar;
  else if (op == PLUS)
    data[actual_index] += scalar;
  else if (op == MINUS)
    data[actual_index] -= scalar;
  else if (op == POW)
    data[actual_index] = pow(data[actual_index], scalar);
}

void 
kamasu_elementwise_array_scalar_1(Op op,
				      float* data, 
				      std::size_t linear_size,
				      const std::size_t* factors, 
				      const int* strides,
				      float scalar)
{
  bd_t bd = gridsize(linear_size);

  kamasu_elementwise_array_scalar_1_knl<<<bd.first, bd.second>>>
    (op, 
     data,
     linear_size,
     factors[0],
     strides[0],
     scalar);
}


#include <resophonic/kamasu/config.hpp>
#include <kernel.h>

// bah bah bah
__global__ void
kamasu_elementwise_array_scalar_2_knl
(Op op,
 float* data,
 unsigned linear_size,
 const std::size_t factor0, const std::size_t factor1,
 const int stride0, const int stride1,
 float scalar)
{
  if (INDEX >= linear_size)
    return;

  unsigned actual_index = 
    INDEX/factor1*stride1 +  unsigned(INDEX % factor1)/factor0*stride0;

  if (op == MULTIPLIES)
    data[actual_index] *= scalar;
  else if (op == DIVIDES)
    data[actual_index] /= scalar;
  else if (op == PLUS)
    data[actual_index] += scalar;
  else if (op == MINUS)
    data[actual_index] -= scalar;
  else if (op == POW)
    data[actual_index] = pow(data[actual_index], scalar);
}

void 
kamasu_elementwise_array_scalar_2(Op op,
				      float* data, 
				      std::size_t linear_size,
				      const std::size_t* factors, 
				      const int* strides,
				      float scalar)
{
  bd_t bd = gridsize(linear_size);

  kamasu_elementwise_array_scalar_2_knl<<<bd.first, bd.second>>>
    (op, 
     data,
     linear_size,
     factors[0],factors[1],
     strides[0],strides[1],
     scalar);
}


#include <resophonic/kamasu/config.hpp>
#include <kernel.h>

// bah bah bah
__global__ void
kamasu_elementwise_array_scalar_3_knl
(Op op,
 float* data,
 unsigned linear_size,
 const std::size_t factor0, const std::size_t factor1, const std::size_t factor2,
 const int stride0, const int stride1, const int stride2,
 float scalar)
{
  if (INDEX >= linear_size)
    return;

  unsigned actual_index = 
    INDEX/factor2*stride2 +  unsigned(INDEX % factor1)/factor0*stride0 +  unsigned(INDEX % factor2)/factor1*stride1;

  if (op == MULTIPLIES)
    data[actual_index] *= scalar;
  else if (op == DIVIDES)
    data[actual_index] /= scalar;
  else if (op == PLUS)
    data[actual_index] += scalar;
  else if (op == MINUS)
    data[actual_index] -= scalar;
  else if (op == POW)
    data[actual_index] = pow(data[actual_index], scalar);
}

void 
kamasu_elementwise_array_scalar_3(Op op,
				      float* data, 
				      std::size_t linear_size,
				      const std::size_t* factors, 
				      const int* strides,
				      float scalar)
{
  bd_t bd = gridsize(linear_size);

  kamasu_elementwise_array_scalar_3_knl<<<bd.first, bd.second>>>
    (op, 
     data,
     linear_size,
     factors[0],factors[1],factors[2],
     strides[0],strides[1],strides[2],
     scalar);
}


#include <resophonic/kamasu/config.hpp>
#include <kernel.h>

// bah bah bah
__global__ void
kamasu_elementwise_array_scalar_4_knl
(Op op,
 float* data,
 unsigned linear_size,
 const std::size_t factor0, const std::size_t factor1, const std::size_t factor2, const std::size_t factor3,
 const int stride0, const int stride1, const int stride2, const int stride3,
 float scalar)
{
  if (INDEX >= linear_size)
    return;

  unsigned actual_index = 
    INDEX/factor3*stride3 +  unsigned(INDEX % factor1)/factor0*stride0 +  unsigned(INDEX % factor2)/factor1*stride1 +  unsigned(INDEX % factor3)/factor2*stride2;

  if (op == MULTIPLIES)
    data[actual_index] *= scalar;
  else if (op == DIVIDES)
    data[actual_index] /= scalar;
  else if (op == PLUS)
    data[actual_index] += scalar;
  else if (op == MINUS)
    data[actual_index] -= scalar;
  else if (op == POW)
    data[actual_index] = pow(data[actual_index], scalar);
}

void 
kamasu_elementwise_array_scalar_4(Op op,
				      float* data, 
				      std::size_t linear_size,
				      const std::size_t* factors, 
				      const int* strides,
				      float scalar)
{
  bd_t bd = gridsize(linear_size);

  kamasu_elementwise_array_scalar_4_knl<<<bd.first, bd.second>>>
    (op, 
     data,
     linear_size,
     factors[0],factors[1],factors[2],factors[3],
     strides[0],strides[1],strides[2],strides[3],
     scalar);
}


#include <resophonic/kamasu/config.hpp>
#include <kernel.h>

// bah bah bah
__global__ void
kamasu_elementwise_array_scalar_5_knl
(Op op,
 float* data,
 unsigned linear_size,
 const std::size_t factor0, const std::size_t factor1, const std::size_t factor2, const std::size_t factor3, const std::size_t factor4,
 const int stride0, const int stride1, const int stride2, const int stride3, const int stride4,
 float scalar)
{
  if (INDEX >= linear_size)
    return;

  unsigned actual_index = 
    INDEX/factor4*stride4 +  unsigned(INDEX % factor1)/factor0*stride0 +  unsigned(INDEX % factor2)/factor1*stride1 +  unsigned(INDEX % factor3)/factor2*stride2 +  unsigned(INDEX % factor4)/factor3*stride3;

  if (op == MULTIPLIES)
    data[actual_index] *= scalar;
  else if (op == DIVIDES)
    data[actual_index] /= scalar;
  else if (op == PLUS)
    data[actual_index] += scalar;
  else if (op == MINUS)
    data[actual_index] -= scalar;
  else if (op == POW)
    data[actual_index] = pow(data[actual_index], scalar);
}

void 
kamasu_elementwise_array_scalar_5(Op op,
				      float* data, 
				      std::size_t linear_size,
				      const std::size_t* factors, 
				      const int* strides,
				      float scalar)
{
  bd_t bd = gridsize(linear_size);

  kamasu_elementwise_array_scalar_5_knl<<<bd.first, bd.second>>>
    (op, 
     data,
     linear_size,
     factors[0],factors[1],factors[2],factors[3],factors[4],
     strides[0],strides[1],strides[2],strides[3],strides[4],
     scalar);
}


