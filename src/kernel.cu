#define BOOST_NO_EXCEPTIONS

#include <resophonic/kamasu/config.hpp>
#include "kernel.h"
#include "cutil.h"
#include "hip/hip_runtime.h"

#include <hipblas.h>

#include "generated/elementwise_array_scalar.cu"
#include "generated/elementwise_array_array.cu"
