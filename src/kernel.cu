#define BOOST_NO_EXCEPTIONS

#include <resophonic/kamasu/config.hpp>
#include "kernel.h"
#include "cutil.h"
#include "hip/hip_runtime.h"

#include <hipblas.h>

#include "elementwise_array_scalar.cu.generated"
#include "elementwise_array_array.cu.generated"
