#include "hip/hip_runtime.h"
#define BOOST_NO_EXCEPTIONS

#include <resophonic/kamasu/config.hpp>
#include "kernel.h"
#include "cutil.h"
#include <stdio.h>
#include <hipblas.h>

#include <boost/preprocessor.hpp>

#define BOOST_PP_ITERATION_LIMITS (1, KAMASU_MAX_ARRAY_DIM-1)
#define BOOST_PP_FILENAME_1 "kernel.ipp"
#include BOOST_PP_ITERATE()

#undef RESOPHONIC_OP
#define RESOPHONIC_OP exp
#define BOOST_PP_ITERATION_LIMITS (1, KAMASU_MAX_ARRAY_DIM-1)
#define BOOST_PP_FILENAME_1 "elementwise_array_op.ipp"
#include BOOST_PP_ITERATE()

#undef RESOPHONIC_OP
#define RESOPHONIC_OP exp2
#define BOOST_PP_ITERATION_LIMITS (1, KAMASU_MAX_ARRAY_DIM-1)
#define BOOST_PP_FILENAME_1 "elementwise_array_op.ipp"
#include BOOST_PP_ITERATE()

#undef RESOPHONIC_OP
#define RESOPHONIC_OP log10
#define BOOST_PP_ITERATION_LIMITS (1, KAMASU_MAX_ARRAY_DIM-1)
#define BOOST_PP_FILENAME_1 "elementwise_array_op.ipp"
#include BOOST_PP_ITERATE()

