#define BOOST_NO_EXCEPTIONS

#include <resophonic/kamasu/config.hpp>
#include "kernel.h"
#include "cutil.h"
#include "hip/hip_runtime.h"

#include <hipblas.h>

#include <boost/preprocessor.hpp>

#define BOOST_PP_ITERATION_LIMITS (1, KAMASU_MAX_ARRAY_DIM-1)
#define BOOST_PP_FILENAME_1 "kernel.ipp"
#include BOOST_PP_ITERATE()

#include "elementwise_array_scalar.cu.generated"
