#include "hip/hip_runtime.h"
#define BOOST_NO_EXCEPTIONS

#include <resophonic/kamasu/config.hpp>
#include <resophonic/kamasu/mirror.hpp>
#include "kernel.h"
#include "cutil.h"
#include <stdio.h>
#include <hipblas.h>
#include <map>


#define SDATA( index)      CUT_BANK_CHECKER(sdata, index)

#include <boost/preprocessor.hpp>

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__device__ float doit(float f)
{
  return f + 1.0f;
}

__global__ void
testKernel( float* g_idata, float* g_odata) 
{
  // shared memory
  // the size is determined by the host application
  extern  __shared__  float sdata[];

  // access thread id
  const unsigned int tid = threadIdx.x +
    + threadIdx.y * blockDim.x 
    + threadIdx.z * blockDim.y * blockDim.x; 

  // read in input data from global memory
  // use the bank checker macro to check for bank conflicts during host
  // emulation
  sdata[tid] = g_idata[tid];
  //__syncthreads();

  // perform some computations
  sdata[tid] = doit(sdata[tid]);
  //__syncthreads();

  // write data to global memory
  g_odata[tid] = sdata[tid];
}

#define X      threadIdx.x  
#define XSIZE  blockDim.x
#define Y      blockIdx.x
#define YSIZE  gridDim.x
#define Z      blockIdx.y
#define ZSIZE  gridDim.y

#define INDEX (Z * YSIZE * XSIZE + Y * XSIZE + X)

__global__ void
gpu_add_elem_knl(float* g_idata1, float* g_idata2, float* g_odata) 
{
  g_odata[INDEX] = g_idata1[INDEX] + g_idata2[INDEX];
}

//
//  vector-scalar knls
//

__global__ void
gpu_add_scalar_k(float* data, float scalar) 
{
  data[INDEX] += scalar;
}

__global__ void
gpu_sub_scalar_k(float* data, float scalar) 
{
  data[INDEX] -= scalar;
}

__global__ void
gpu_mul_scalar_k(float* data, float scalar) 
{
  data[INDEX] *= scalar;
}

__global__ void
gpu_div_scalar_k(float* data, float scalar) 
{
  data[INDEX] /= scalar;
}


//
//  c-iface scalar
//

void gpu_mul_scalar(float* data, unsigned size, float scalar)
{
  hipblasSscal(size, scalar, data, 1);
  //  gpu_mul_scalar_k<<<gridsize(size), threads_per_block>>>(data, scalar);
}

void gpu_add_scalar(float* data, unsigned size, float scalar)
{
  bd_t bd = gridsize(size);
  gpu_add_scalar_k<<<bd.first, bd.second>>>(data, scalar);
}

void gpu_sub_scalar(float* data, unsigned size, float scalar)
{
  bd_t bd = gridsize(size);
  gpu_sub_scalar_k<<<bd.first, bd.second>>>(data, scalar);
}

void gpu_div_scalar(float* data, unsigned size, float scalar)
{
  hipblasSscal(size, 1.0f/scalar, data, 1);
}

//
//  vector-vector knls
//

__global__ void
gpu_add_vector_k(float* lhs, float* rhs) 
{
  lhs[INDEX] += rhs[INDEX];
}

__global__ void
gpu_sub_vector_k(float* lhs, float* rhs) 
{
  lhs[INDEX] -= rhs[INDEX];
}

__global__ void
gpu_mul_vector_k(float* lhs, float* rhs) 
{
  lhs[INDEX] *= rhs[INDEX];
}

__global__ void
gpu_div_vector_k(float* lhs, float* rhs) 
{
  lhs[INDEX] /= rhs[INDEX];
}

__global__ void
kamasu_testy_knl_thunk(Op op,
		       float* data,
		       unsigned linear_size,
		       unsigned n_dims,
		       unsigned* factors,
		       int* strides,
		       float scalar)
{
  unsigned index = INDEX;
  unsigned actual_index = 0;

  if (INDEX >= linear_size)
    return;

  for (int i = n_dims-1; i>=0; i--)
    {
      unsigned this_component = index / factors[i];
      actual_index += this_component * strides[i];
      index -= this_component * factors[i];
    }

  if (op == MULTIPLIES)
    data[actual_index] *= scalar;
  else if (op == DIVIDES)
    data[actual_index] /= scalar;
  else if (op == PLUS)
    data[actual_index] += scalar;
  else if (op == MINUS)
    data[actual_index] -= scalar;
  else if (op == POW)
    data[actual_index] = pow(data[actual_index], scalar);
}

#define BOOST_PP_ITERATION_LIMITS (1, KAMASU_MAX_ARRAY_DIM-1)
#define BOOST_PP_FILENAME_1 "kernel.ipp"
#include BOOST_PP_ITERATE()


void kamasu_testy_knl(Op op,
		      float* data, 
		      std::size_t linear_size,
		      unsigned n_dims, 
		      unsigned* factors, int* strides, 
		      float scalar)
{
  bd_t bd = gridsize(linear_size);
  //printf("%s with gridsize %u %u\n", __PRETTY_FUNCTION__, bd.first, bd.second);
  //printf("n_dims=%u linear_size=%zu", n_dims, linear_size);
  
  kamasu_testy_knl_thunk<<<bd.first, bd.second>>>(op, 
						  data,
						  linear_size,
						  n_dims,
						  factors,
						  strides,
						  scalar);
}


#define SCALAR_KNL(Z, N, DATA)						\
  __global__ void							\
  gpu_add_scalar_knl_ ## N						\
  (float* data, unsigned offset,					\
   BOOST_PP_ENUM_PARAMS(N, unsigned dim),				\
   BOOST_PP_ENUM_PARAMS(N, unsigned factor),				\
   BOOST_PP_ENUM_PARAMS(N, unsigned stride),				\
   float scalar)							\
  {									\
    unsigned index = INDEX;						\
    unsigned r = 0;							\
  }


