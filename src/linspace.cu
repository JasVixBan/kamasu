#include "hip/hip_runtime.h"

#include "kernel_util.hpp"

__global__ void
kamasu_linspace_knl(float* data,
		    unsigned linear_size,
		    const int stride,
		    float start,
		    float stop)
{
  if (INDEX >= linear_size)
    return;

  unsigned actual_index =  INDEX*stride;

  data[actual_index] = start + INDEX * (stop - start) / (linear_size - 1);
}

void 
kamasu_linspace(float* data, 
		std::size_t linear_size,
		const int stride,
		float start,
		float stop)
{
  bd_t bd = gridsize(linear_size);

  kamasu_linspace_knl<<<bd.first, bd.second>>>
    (data,
     linear_size,
     stride,
     start,
     stop);
}

