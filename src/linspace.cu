#include "hip/hip_runtime.h"

#include "kernel_util.hpp"

template <typename T>
__global__ void
kamasu_linspace_knl(T* data,
		    unsigned linear_size,
		    const int stride,
		    T start,
		    T stop)
{
  if (INDEX >= linear_size)
    return;

  unsigned actual_index =  INDEX*stride;

  data[actual_index] = start + INDEX * (stop - start) / (linear_size - 1);
}

template <typename T>
void 
kamasu_linspace(T* data, 
		std::size_t linear_size,
		const int stride,
		T start,
		T stop)
{
  bd_t bd = gridsize(linear_size);

  kamasu_linspace_knl<<<bd.first, bd.second>>>
    (data,
     linear_size,
     stride,
     start,
     stop);
}

template void kamasu_linspace<float>(float*, std::size_t, const int, float, float);
