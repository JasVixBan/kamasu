#include "hip/hip_runtime.h"
#include "kernel_util.hpp"

namespace resophonic 
{
  namespace kamasu 
  {

    template <typename T>
    __global__ void
    linspace_knl(T* data,
			unsigned linear_size,
			const int stride,
			T start,
			T stop)
    {
      if (INDEX >= linear_size)
	return;

      unsigned actual_index =  INDEX*stride;

      data[actual_index] = start + INDEX * (stop - start) / (linear_size - 1);
    }

    template <typename T>
    void 
    linspace(T* data, 
		    std::size_t linear_size,
		    const int stride,
		    T start,
		    T stop)
    {
      bd_t bd = gridsize(linear_size);

      linspace_knl<<<bd.first, bd.second>>>
	(data,
	 linear_size,
	 stride,
	 start,
	 stop);
    }

    template void linspace<float>(float*, std::size_t, const int, float, float);
  }
}
