#include "hip/hip_runtime.h"

#include "transform.hpp"
#include "kernel_util.hpp"

#include <proto_tags_fwd.hpp>
#include <resophonic/kamasu/tag.hpp>
#include "primitives.hpp"

namespace resophonic {
  namespace kamasu {

    //
    // array-scalar
    //  
    template <typename T, int N, typename Tag>
    __global__ void 
    transform_knl(T* data, 
	    std::size_t linear_size, 
	    argpack<std::size_t,N> factors,
	    argpack<int, N> strides,
	    T scalar)
    {
      if (INDEX >= linear_size)
	return;

      unsigned thisthread_offset = actual_index<N>(factors, strides);

      op_impl_<T, Tag>::impl(data + thisthread_offset, scalar); 
    }

    template <typename T, int N, typename Tag>
    void 
    transform(T* data, 
	      std::size_t linear_size, 
	      const std::size_t* factors,
	      const int* strides,
	      T scalar)
    {
      bd_t bd = gridsize(linear_size);
      
      argpack<std::size_t, N> factors_(factors);
      argpack<int, N> strides_(strides);
      
      transform_knl<T, N, Tag><<<bd.first, bd.second>>>(data, linear_size, factors_, strides_, scalar);
    }

    namespace inst {
      template <typename T, typename Tag>
      struct eas 
      {
	eas()
	{
	  transform<T, 1, Tag>(0, 0, 0, 0, 0.0f);
	  transform<T, 2, Tag>(0, 0, 0, 0, 0.0f);
	  transform<T, 3, Tag>(0, 0, 0, 0, 0.0f);
	  transform<T, 4, Tag>(0, 0, 0, 0, 0.0f);
	  transform<T, 5, Tag>(0, 0, 0, 0, 0.0f);
	}
      };

      template struct eas<float, boost::proto::tag::plus>;
      template struct eas<float, boost::proto::tag::minus>;
      template struct eas<float, boost::proto::tag::multiplies>;
      template struct eas<float, boost::proto::tag::divides>;
      template struct eas<float, boost::proto::tag::assign>;
      template struct eas<float, resophonic::kamasu::tag::pow>;
    }

    //
    // array-array
    //
    template <typename T, int N, typename Tag>
    __global__ void 
    transform_knl(std::size_t linear_size,
		  T* data_l,
		  T* data_r,
		  argpack<std::size_t, N> factors_l,
		  argpack<std::size_t, N> factors_r,
		  argpack<int, N> strides_l,
		  argpack<int, N> strides_r)
    {
      if (INDEX >= linear_size)
	return;

      unsigned lhs_off = actual_index<N>(factors_l, strides_l);
      unsigned rhs_off = actual_index<N>(factors_r, strides_r);

      op_impl_<T, Tag>::impl(data_l + lhs_off, data_r + rhs_off); 
    }

    template <typename T, int N, typename Tag>
    void 
    transform(std::size_t linear_size,
	      T* data_l,
	      T* data_r,
	      const std::size_t* factors_l,
	      const std::size_t* factors_r,
	      const int* strides_l,
	      const int* strides_r,
	      hipStream_t stream)
    {
      bd_t bd = gridsize(linear_size);
      
      argpack<std::size_t, N> factors_l_(factors_l);
      argpack<int, N> strides_l_(strides_l);
      
      argpack<std::size_t, N> factors_r_(factors_r);
      argpack<int, N> strides_r_(strides_r);
      
      transform_knl<T, N, Tag><<<bd.first, bd.second, 0, stream>>>(linear_size, 
								   data_l, data_r, 
								   factors_l_, factors_r_,
								   strides_l_, strides_r_);
    }

    namespace inst 
    {
      template <typename T, typename Tag>
      struct iaao
      {
	iaao()
	{
	  transform<T, 1, Tag>(0, 0, 0, 0, 0, 0, 0, 0);
	  transform<T, 2, Tag>(0, 0, 0, 0, 0, 0, 0, 0);
	  transform<T, 3, Tag>(0, 0, 0, 0, 0, 0, 0, 0);
	  transform<T, 4, Tag>(0, 0, 0, 0, 0, 0, 0, 0);
	  transform<T, 5, Tag>(0, 0, 0, 0, 0, 0, 0, 0);
	}
      };

      template struct iaao<float, boost::proto::tag::plus>;
      template struct iaao<float, boost::proto::tag::minus>;
      template struct iaao<float, boost::proto::tag::divides>;

    }



  }
}


