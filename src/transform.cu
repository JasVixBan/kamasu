#include "hip/hip_runtime.h"

#include "transform.hpp"
#include "kernel_util.hpp"

#include <iostream>
#include <proto_tags_fwd.hpp>
#include <resophonic/kamasu/tag.hpp>
#include "primitives.hpp"

#include <resophonic/kamasu/array.hpp>


namespace resophonic {
  namespace kamasu {

    //
    // array-scalar
    //  
    template <typename T, typename Tag>
    __global__ void 
    transform_knl(T* data, view_params vp, T scalar)
    {
      if (INDEX >= vp.linear_size)
	return;

      unsigned thisthread_offset = actual_index(vp.nd, vp.factors, vp.strides);

      op_impl_<T, Tag>::impl(data + thisthread_offset, scalar); 
    }

    template <typename T, typename Tag>
    void 
    transform(T* data, const view_params& vp,
	      T scalar)
    {
      bd_t bd = gridsize(vp.linear_size);
      
      std::cout << "launch: " << bd.first << " " << bd.second << "\n"
		<< "nd = " << vp.nd << " factor0=" << vp.factors[0]
		<< " stride0 = " << vp.strides[0] << "\n";
      
      transform_knl<T, Tag><<<bd.first, bd.second>>>(data + vp.offset, vp, scalar);
    }

    namespace inst {
      template <typename T, typename Tag>
      struct eas 
      {
	eas()
	{
	  view_params vp;
	  transform<T, Tag>(0, vp, 0.0f);
	}
      };

      template struct eas<float, boost::proto::tag::plus>;
      template struct eas<float, boost::proto::tag::minus>;
      template struct eas<float, boost::proto::tag::multiplies>;
      template struct eas<float, boost::proto::tag::divides>;
      template struct eas<float, boost::proto::tag::assign>;
      template struct eas<float, resophonic::kamasu::tag::pow>;
    }

    //
    // array-array
    //
    template <typename T, int N, typename Tag>
    __global__ void 
    transform_knl(std::size_t linear_size,
		  T* data_l,
		  T* data_r,
		  argpack<std::size_t, N> factors_l,
		  argpack<std::size_t, N> factors_r,
		  argpack<int, N> strides_l,
		  argpack<int, N> strides_r)
    {
      if (INDEX >= linear_size)
	return;

      unsigned lhs_off = actual_index<N>(factors_l, strides_l);
      unsigned rhs_off = actual_index<N>(factors_r, strides_r);

      op_impl_<T, Tag>::impl(data_l + lhs_off, data_r + rhs_off); 
    }

    template <typename T, int N, typename Tag>
    void 
    transform(std::size_t linear_size,
	      T* data_l,
	      T* data_r,
	      const std::size_t* factors_l,
	      const std::size_t* factors_r,
	      const int* strides_l,
	      const int* strides_r,
	      hipStream_t stream)
    {
      bd_t bd = gridsize(linear_size);
      
      argpack<std::size_t, N> factors_l_(factors_l);
      argpack<int, N> strides_l_(strides_l);
      
      argpack<std::size_t, N> factors_r_(factors_r);
      argpack<int, N> strides_r_(strides_r);
      
      transform_knl<T, N, Tag><<<bd.first, bd.second, 0, stream>>>(linear_size, 
								   data_l, data_r, 
								   factors_l_, factors_r_,
								   strides_l_, strides_r_);
    }

    namespace inst 
    {
      template <typename T, typename Tag>
      struct iaao
      {
	iaao()
	{
	  transform<T, 1, Tag>(0, 0, 0, 0, 0, 0, 0, 0);
	  transform<T, 2, Tag>(0, 0, 0, 0, 0, 0, 0, 0);
	  transform<T, 3, Tag>(0, 0, 0, 0, 0, 0, 0, 0);
	  transform<T, 4, Tag>(0, 0, 0, 0, 0, 0, 0, 0);
	  transform<T, 5, Tag>(0, 0, 0, 0, 0, 0, 0, 0);
	}
      };

      template struct iaao<float, boost::proto::tag::plus>;
      template struct iaao<float, boost::proto::tag::minus>;
      template struct iaao<float, boost::proto::tag::divides>;

    }



  }
}


