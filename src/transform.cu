#include "hip/hip_runtime.h"

#include "transform.hpp"
#include "kernel_util.hpp"

#include <iostream>
#include <proto_tags_fwd.hpp>
#include <resophonic/kamasu/tag.hpp>
#include "primitives.hpp"

#include <resophonic/kamasu/array.hpp>
#include <resophonic/kamasu/exception.hpp>


namespace resophonic {
  namespace kamasu {

    //
    // array-scalar
    //  
    template <typename T, typename Tag>
    __global__ void 
    transform_knl(T* data, view_params vp, T scalar)
    {
      if (INDEX >= vp.linear_size)
	return;

      unsigned thisthread_offset = actual_index(INDEX, vp.nd, vp.factors, vp.strides);

      op_impl_<T, Tag>::impl(data + thisthread_offset, scalar); 
    }

    template <typename T, typename Tag>
    void 
    transform(T* data, const view_params& vp,
	      T scalar)
    {
      bd_t bd = gridsize(vp.linear_size);
      
      std::cout << "launch: " << bd.first << " " << bd.second << "\n"
		<< "nd = " << vp.nd << " factor0=" << vp.factors[0]
		<< " stride0 = " << vp.strides[0] << "\n";
      
      transform_knl<T, Tag><<<bd.first, bd.second>>>(data + vp.offset, vp, scalar);
      cuda_check();
    }

#define INST(TYPE, TAG)							\
    template void transform<TYPE, TAG>(TYPE*, const view_params&, TYPE);

    INST(float, boost::proto::tag::plus);
    INST(float, boost::proto::tag::minus);
    INST(float, boost::proto::tag::multiplies);
    INST(float, boost::proto::tag::divides);
    INST(float, boost::proto::tag::assign);
    INST(float, resophonic::kamasu::tag::pow);

#undef INST

    //
    // array-array
    //
    template <typename T, typename Tag>
    __global__ void 
    transform_knl(T* data_l, view_params vp_l,
		  T* data_r, view_params vp_r)
    {
      if (INDEX >= vp_l.linear_size)
	return;

      unsigned lhs_off = actual_index(INDEX, vp_l.nd, vp_l.factors, vp_l.strides);
      unsigned rhs_off = actual_index(INDEX, vp_r.nd, vp_r.factors, vp_r.strides);

      op_impl_<T, Tag>::impl(data_l + lhs_off, data_r + rhs_off); 
    }

    template <typename T, typename Tag>
    void 
    transform(T* data_l, const view_params& vp_l,
	      T* data_r, const view_params& vp_r)
    {
      bd_t bd = gridsize(vp_l.linear_size, 16);
      
      transform_knl<T, Tag><<<bd.first, bd.second>>>(data_l + vp_l.offset, vp_l, 
						     data_r + vp_r.offset, vp_r); 
      cuda_check();
    }

#define INST(TYPE, TAG)							\
    template void transform<TYPE, TAG>(TYPE*, const view_params&,	\
				       TYPE*, const view_params&);

    INST(float, boost::proto::tag::plus);
    INST(float, boost::proto::tag::minus);
    INST(float, boost::proto::tag::divides);

#undef INST




  }
}


