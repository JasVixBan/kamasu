#include "hip/hip_runtime.h"
#include "kernel_util.hpp"

namespace resophonic {
  namespace kamasu {

    __global__ void
    kamasu_elementwise_array_array_knl_/*N*/(Op op,
					     std::size_t linear_size,
					     float* data_l,
					     float* data_r,
					     /*enum('const std::size_t factor_l%d', N)*/,
					     /*enum('const std::size_t factor_r%d', N)*/,
					     /*enum('const int stride_l%d', N)*/,
					     /*enum('const int stride_r%d', N)*/)
    {
      if (INDEX >= linear_size)
	return;

      unsigned actual_index_l = /* ' + '.join(['INDEX/factor_l%d*stride_l%d' % (N-1, N-1)]+[' unsigned(INDEX %% factor_l%d)/factor_l%d*stride_l%d' % (n+1,n,n) for n in range(N-1)]) */;
      unsigned actual_index_r = /* ' + '.join(['INDEX/factor_r%d*stride_r%d' % (N-1, N-1)]+[' unsigned(INDEX %% factor_r%d)/factor_r%d*stride_r%d' % (n+1,n,n) for n in range(N-1)]) */;

      if (op == MULTIPLIES)
	data_l[actual_index_l] *= data_r[actual_index_r];
      else if (op == DIVIDES)
	data_l[actual_index_l] /= data_r[actual_index_r];
      else if (op == PLUS)
	data_l[actual_index_l] += data_r[actual_index_r];
      else if (op == MINUS)
	data_l[actual_index_l] -= data_r[actual_index_r];
    }

    void 
    kamasu_elementwise_array_array_/*N*/(Op op,
					 std::size_t linear_size,
					 float* data_l,
					 float* data_r,
					 const std::size_t* factors_l,
					 const std::size_t* factors_r,
					 const int* strides_l,
					 const int* strides_r,
					 hipStream_t stream)
    {
      bd_t bd = gridsize(linear_size);
  
      kamasu_elementwise_array_array_knl_/*N*/<<<bd.first, bd.second, 0, stream>>>
	(op, 
	 linear_size,
	 data_l,
	 data_r,
	 /*enum('factors_l[%d]', N)*/,
	 /*enum('factors_r[%d]', N)*/,
	 /*enum('strides_l[%d]', N)*/,
	 /*enum('strides_r[%d]', N)*/);
    }

  }
}
