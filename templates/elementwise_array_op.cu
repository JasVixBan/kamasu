#include "hip/hip_runtime.h"
#include "kernel.h"
#include "cutil.h"

//
//
//  ops
//
//  

__global__ void
kamasu_elementwise_array_/*OP*/_/*N*/_knl
(float* data,
 unsigned linear_size,
 /*', '.join(['const std::size_t factor%d' % x for x in range(N)])*/,
 /*', '.join(['const int stride%d' % x for x in range(N)])*/)
{
  if (INDEX >= linear_size)
    return;

  unsigned actual_index = /* ' + '.join([' unsigned(INDEX/factor%d)*stride%d' % (n,n) for n in range(N)]) */;

  data[actual_index] = /*OP*/(data[actual_index]);
}

void 
BOOST_PP_CAT(kamasu_elementwise_array_/*OP*/_/*N*/(float* data, 
							std::size_t linear_size,
							const std::size_t* factors, 
							const int* strides)
{
  bd_t bd = gridsize(linear_size);

  kamasu_elementwise_array_/*OP*/_/*N*/_knl<<<bd.first, bd.second>>>
    (data,
     linear_size,
     /*','.join(['factors[%d]' % x for x in range(N)])*/,
     /*','.join(['strides[%d]' % x for x in range(N)])*/);
}

