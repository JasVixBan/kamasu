#include "hip/hip_runtime.h"
__global__ void
kamasu_elementwise_array_scalar_/*N*/_knl
(Op op,
 float* data,
 unsigned linear_size,
 /*', '.join(['const std::size_t factor%d' % x for x in range(N)])*/,
 /*', '.join(['const int stride%d' % x for x in range(N)])*/,
 float scalar)
{
  if (INDEX >= linear_size)
    return;

  unsigned actual_index = 
    /* ' + '.join(['INDEX/factor%d*stride%d' % (N-1, N-1)]+[' unsigned(INDEX %% factor%d)/factor%d*stride%d' % (n+1,n,n) for n in range(N-1)]) */;

  if (op == MULTIPLIES)
    data[actual_index] *= scalar;
  else if (op == DIVIDES)
    data[actual_index] /= scalar;
  else if (op == PLUS)
    data[actual_index] += scalar;
  else if (op == MINUS)
    data[actual_index] -= scalar;
  else if (op == POW)
    data[actual_index] = pow(data[actual_index], scalar);
}

void 
kamasu_elementwise_array_scalar_/*N*/(Op op,
				      float* data, 
				      std::size_t linear_size,
				      const std::size_t* factors, 
				      const int* strides,
				      float scalar)
{
  bd_t bd = gridsize(linear_size);

  kamasu_elementwise_array_scalar_/*N*/_knl<<<bd.first, bd.second>>>
    (op, 
     data,
     linear_size,
     /*','.join(['factors[%d]' % x for x in range(N)])*/,
     /*','.join(['strides[%d]' % x for x in range(N)])*/,
     scalar);
}

