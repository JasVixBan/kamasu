#include "hip/hip_runtime.h"
#include <resophonic/kamasu/config.hpp>
#include "kernel.h"

__global__ void
kamasu_unary_array_/*OP*/_/*N*/_knl
(float* data,
 unsigned linear_size,
 /*', '.join(['const std::size_t factor%d' % x for x in range(N)])*/,
 /*', '.join(['const int stride%d' % x for x in range(N)])*/)
{
  if (INDEX >= linear_size)
    return;

  unsigned actual_index = /* ' + '.join([' unsigned(INDEX/factor%d)*stride%d' % (n,n) for n in range(N)]) */;

  data[actual_index] = /*OP*/(data[actual_index]);
}

void 
kamasu_unary_array_/*OP*/_/*N*/(float* data, 
				std::size_t linear_size,
				const std::size_t* factors, 
				const int* strides)
{
  bd_t bd = gridsize(linear_size);

  kamasu_unary_array_/*OP*/_/*N*/_knl<<<bd.first, bd.second>>>
    (data,
     linear_size,
     /*','.join(['factors[%d]' % x for x in range(N)])*/,
     /*','.join(['strides[%d]' % x for x in range(N)])*/);
}

