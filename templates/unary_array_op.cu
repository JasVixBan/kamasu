#include "hip/hip_runtime.h"
#include "kernel_util.hpp"
#include <resophonic/kamasu/view_params.hpp>

namespace resophonic {
  namespace kamasu {

    __global__ void
    unary_array_/*OP*/_/*N*/_knl
    (float* data,
     unsigned linear_size,
     /*', '.join(['const factor_t factor%d' % x for x in range(N)])*/,
     /*', '.join(['const stride_t stride%d' % x for x in range(N)])*/)
    {
      if (INDEX >= linear_size)
	return;

      unsigned actual_index = /* ' + '.join([' unsigned(INDEX/factor%d)*stride%d' % (n,n) for n in range(N)]) */;

      data[actual_index] = /*OP*/(data[actual_index]);
    }

    void 
    unary_array_/*OP*/_/*N*/(float* data, 
				    std::size_t linear_size,
				    const factor_t* factors, 
				    const stride_t* strides,
				    hipStream_t stream)
    {
      bd_t bd = gridsize(linear_size);

      unary_array_/*OP*/_/*N*/_knl<<<bd.first, bd.second, 0, stream>>>
	(data,
	 linear_size,
	 /*','.join(['factors[%d]' % x for x in range(N)])*/,
	 /*','.join(['strides[%d]' % x for x in range(N)])*/);
    }

  }
}
