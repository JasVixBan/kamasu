#include "hip/hip_runtime.h"
#include <resophonic/kamasu/array_impl.hpp>
#include <resophonic/kamasu/view_params.hpp>

#include "kernel_util.hpp"


namespace resophonic
{

  namespace kamasu
  {
    //
    // array-scalar
    //  
    __global__ void 
    kuhkuhknl(view_params a)
    {
      a.nd = 2;
      /*
      if (INDEX >= linear_size)
	return;

      unsigned thisthread_offset = actual_index<N>(factors, strides);

      data[thisthread_offset] = data[thisthread_offset] * data[thisthread_offset];
      */
    }

    template <typename T>
    void 
    bang(array_impl<T> a)
    {
      bd_t bd = gridsize(a.linear_size());
      
      // argpack<std::size_t, N> factors_(factors);
      //      //      argpack<int, N> strides_(strides);
      //      knl<T, N, Tag><<<bd.first, bd.second>>>(data, linear_size, factors_, strides_, scalar);
      //      blonk b;
      view_params vp;

      kuhkuhknl<<<bd.first, bd.second>>>(vp);
    }

    template void bang(array_impl<float>);

  }
}

